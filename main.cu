#include <hip/hip_runtime.h>
#include <cstdio>

#include "matmulf8.cuh"


int main() {
    int n = 1024, m = 1024, p = 1024;
    int *A, *B, *C;
    hipSetDevice(0);
    hipFree(0);
    
    hipHostMalloc(&A, n * m * sizeof(int) / 4);
    hipHostMalloc(&B, m * p * sizeof(int) / 4);
    hipHostMalloc(&C, n * p * sizeof(int) / 4);
    for(int i = 0; i < n * m / 4; i++) {
        A[i] = rand();
        // A[i] &= 0x7f7f7f7f;
        A[i] = 0;
    }
    for(int i = 0; i < m * p / 4; i++) {
        B[i] = rand();
        // B[i] &= 0x7f7f7f7f;
        B[i] = 0;
    }
    float t = matmul(A, B, C, n, m, p);
    printf("Time: %f ms\n", t);
    float flops = 2.0 * n * m * p / t / 1e6;
    printf("FLOPS: %f GFLOPS\n", flops);
    hipHostFree(A); hipHostFree(B); hipHostFree(C);
    return 0;
}