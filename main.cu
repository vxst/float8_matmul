#include <hip/hip_runtime.h>
#include <cstdio>

#include "matmulf8.cuh"
#include "load_core.cuh"


int main() {
    int n = 4096, m = 4096, p = 4096;
    int *A, *B, *C;
    hipSetDevice(0);
    hipFree(0);
    
    hipHostMalloc(&A, n * m * sizeof(int) / 4);
    hipHostMalloc(&B, m * p * sizeof(int) / 4);
    hipHostMalloc(&C, n * p * sizeof(int) / 4);
#ifdef DB
    int* acore = load_core("cores/f8e5m2_adbcore.bin");
#else
    int* acore = load_core("cores/f8e5m2_acore.bin");
#endif
    int* mcore = load_core("cores/f8e5m2_mcore.bin");
    for(int i = 0; i < n * m / 4; i++) {
        A[i] = rand();
        // A[i] &= 0x7f7f7f7f;
        A[i] = 0;
    }
    for(int i = 0; i < m * p / 4; i++) {
        B[i] = rand();
        // B[i] &= 0x7f7f7f7f;
        B[i] = 0;
    }
    float t = matmul(A, B, C, n, m, p, acore, mcore);
    printf("Time: %f ms\n", t);
    float flops = 2.0 * n * m * p / t / 1e6;
    printf("FLOPS: %f GFLOPS\n", flops);
    hipHostFree(A); hipHostFree(B); hipHostFree(C);
    hipHostFree(acore); hipHostFree(mcore);
    return 0;
}