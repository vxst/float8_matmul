#include <hip/hip_runtime.h>
#include "matmulf8_kernel.cuh"

float matmul(int* A, int* B, int* C, int n, int m, int p, int* acore, int* mcore) {
    int* d_A, *d_B, *d_C, *d_acore, *d_mcore;
    hipMalloc(&d_A, n * m / 4 * sizeof(int));
    hipMalloc(&d_B, m * p / 4 * sizeof(int));
    hipMalloc(&d_C, n * p / 4 * sizeof(int));
    hipMalloc(&d_acore, 16384);
    hipMalloc(&d_mcore, 16384);
    hipMemcpy(d_A, A, n * m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, m * p * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_acore, acore, 16384, hipMemcpyHostToDevice);
    hipMemcpy(d_mcore, mcore, 16384, hipMemcpyHostToDevice);

    float t;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // Each kernel thread do 4xf8 result
    matmulf8<<<dim3(n / 32, p / 32), dim3(32, 8)>>>(d_A, d_B, d_C, n, m, p, d_acore, d_mcore);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t, start, stop);

    hipMemcpy(C, d_C, n * p * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_acore);
    hipFree(d_mcore);

    return t;
}


