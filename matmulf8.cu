// Copyright (C) 2024 Chunqing Shan
// 
// float8_matmul is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// float8_matmul is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU Lesser General Public License for more details.
// 
// You should have received a copy of the GNU Lesser General Public License
// along with float8_matmul. If not, see <http://www.gnu.org/licenses/>.

#include <cstdio>

#include <hip/hip_runtime.h>
#include "load_core.cuh"
#include "matmulf8_kernel.cuh"

float matmul(int* A, int* B, int* C, int n, int m, int p, int* mcore) {
    int* d_A, *d_B, *d_C, *d_mcore;
    hipMalloc(&d_A, n * m / 4 * sizeof(int));
    hipMalloc(&d_B, m * p / 4 * sizeof(int));
    hipMalloc(&d_C, n * p / 4 * sizeof(int));
    hipMalloc(&d_mcore, 16384);
    hipMemcpy(d_A, A, n * m * sizeof(int) / 4, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, m * p * sizeof(int) / 4, hipMemcpyHostToDevice);
    hipMemcpy(d_mcore, mcore, 16384, hipMemcpyHostToDevice);

    float t;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // Each kernel thread do 4xf8 result
    matmulf8<<<dim3(n / 32, p / 32), dim3(32, 8)>>>(d_A, d_B, d_C, n, m, p, d_mcore);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t, start, stop);

    hipMemcpy(C, d_C, n * p * sizeof(int) / 4, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_mcore);

    return t;
}
