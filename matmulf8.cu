#include <cstdio>

#include <hip/hip_runtime.h>
#include "load_core.cuh"
#include "matmulf8_kernel.cuh"

float matmul(int* A, int* B, int* C, int n, int m, int p, int* acore, int* mcore) {
    int* d_A, *d_B, *d_C, *d_acore, *d_mcore;
    hipMalloc(&d_A, n * m / 4 * sizeof(int));
    hipMalloc(&d_B, m * p / 4 * sizeof(int));
    hipMalloc(&d_C, n * p / 4 * sizeof(int));
    hipMalloc(&d_acore, 16384);
    hipMalloc(&d_mcore, 16384);
    hipMemcpy(d_A, A, n * m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, m * p * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_acore, acore, 16384, hipMemcpyHostToDevice);
    hipMemcpy(d_mcore, mcore, 16384, hipMemcpyHostToDevice);

    float t;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // Each kernel thread do 4xf8 result
    matmulf8<<<dim3(n / 32, p / 32), dim3(32, 8)>>>(d_A, d_B, d_C, n, m, p, d_acore, d_mcore);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t, start, stop);

    hipMemcpy(C, d_C, n * p * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_acore);
    hipFree(d_mcore);

    return t;
}

int main() {
    int n = 1024, m = 1024, p = 1024;
    int *A, *B, *C;
    hipHostMalloc(&A, n * m * sizeof(int) / 4);
    hipHostMalloc(&B, m * p * sizeof(int) / 4);
    hipHostMalloc(&C, n * p * sizeof(int) / 4);
    int* acore = load_core("addcore.bin");
    int* mcore = load_core("mltcore.bin");
    for(int i = 0; i < n * m; i++) {
        A[i] = rand() & 0xff;
    }
    for(int i = 0; i < m * p; i++) {
        B[i] = rand() & 0xff;
    }
    float t = matmul(A, B, C, n, m, p, acore, mcore);
    printf("Time: %f ms\n", t);
    hipHostFree(A); hipHostFree(B); hipHostFree(C);
    hipHostFree(acore); hipHostFree(mcore);
    return 0;
}