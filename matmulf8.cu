// Copyright (C) 2024 Chunqing Shan
// 
// float8_matmul is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// float8_matmul is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU Lesser General Public License for more details.
// 
// You should have received a copy of the GNU Lesser General Public License
// along with float8_matmul. If not, see <http://www.gnu.org/licenses/>.

#include <cstdio>

#include <hip/hip_runtime.h>
#include "load_core.cuh"
#include "matmulf8_kernel.cuh"

float matmul(int* A, int* B, int* C, int n, int m, int p, int* acore, int* mcore) {
    int* d_A, *d_B, *d_C, *d_acore, *d_mcore;
    hipMalloc(&d_A, n * m / 4 * sizeof(int));
    hipMalloc(&d_B, m * p / 4 * sizeof(int));
    hipMalloc(&d_C, n * p / 4 * sizeof(int));
    hipMalloc(&d_acore, 16384);
    hipMalloc(&d_mcore, 16384);
    hipMemcpy(d_A, A, n * m * sizeof(int) / 4, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, m * p * sizeof(int) / 4, hipMemcpyHostToDevice);
    hipMemcpy(d_acore, acore, 16384, hipMemcpyHostToDevice);
    hipMemcpy(d_mcore, mcore, 16384, hipMemcpyHostToDevice);

    float t;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // Each kernel thread do 4xf8 result
    matmulf8<<<dim3(n / 32, p / 32), dim3(32, 8)>>>(d_A, d_B, d_C, n, m, p, d_acore, d_mcore);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t, start, stop);

    hipMemcpy(C, d_C, n * p * sizeof(int) / 4, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_acore);
    hipFree(d_mcore);

    return t;
}

int main() {
    int n = 4096, m = 4096, p = 4096;
    int *A, *B, *C;
    hipSetDevice(0);
    hipFree(0);
    
    hipHostMalloc(&A, n * m * sizeof(int) / 4);
    hipHostMalloc(&B, m * p * sizeof(int) / 4);
    hipHostMalloc(&C, n * p * sizeof(int) / 4);
#ifdef DB
    int* acore = load_core("addcore.bin");
#else
    int* acore = load_core("apdcore.bin");
#endif
    int* mcore = load_core("mltcore.bin");
    for(int i = 0; i < n * m / 4; i++) {
        A[i] = rand();
        // A[i] &= 0x7f7f7f7f;
        A[i] = 0;
    }
    for(int i = 0; i < m * p / 4; i++) {
        B[i] = rand();
        // B[i] &= 0x7f7f7f7f;
        B[i] = 0;
    }
    float t = matmul(A, B, C, n, m, p, acore, mcore);
    printf("Time: %f ms\n", t);
    float flops = 2.0 * n * m * p / t / 1e6;
    printf("FLOPS: %f GFLOPS\n", flops);
    hipHostFree(A); hipHostFree(B); hipHostFree(C);
    hipHostFree(acore); hipHostFree(mcore);
    return 0;
}