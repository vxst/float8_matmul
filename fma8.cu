// Copyright (C) 2024 Chunqing Shan
// 
// f8matmul is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// qrand is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU Lesser General Public License for more details.
// 
// You should have received a copy of the GNU Lesser General Public License
// along with qrand. If not, see <http://www.gnu.org/licenses/>.


#include <hip/hip_runtime.h>


__device__ __forceinline__ int access_byte(int* __restrict__ data, int i) {
    return data[i>>2] >> ((i&3)<<3) & 0xff;
}

// Do a 4 8bit fma, r[i] = a[i] * b[i] + c[i]
__device__ __forceinline__ int fma8v4(int a, int b, int c, int* __restrict__ acore, int* __restrict__ mcore) {
    int res = 0;
#pragma unroll
    for(int i = 0; i < 4; i++) {
        int a0 = (a >> (i * 8)) & 0xff;
        int b0 = (b >> (i * 8)) & 0xff;
        int c0 = (c >> (i * 8)) & 0xff;
        int m = access_byte(mcore, ((a0&0x7f)<<7) + (b0&0x7f));
        int r = 0;
        m |= (a0&0x80) ^ (b0&0x80);
        if((m&0x80)^(c0&0x80)) {
            if((m&0x7f) == (c0&0x7f)){
                r = 0x00;
            }else if(m&0x80){
                // c0 - m
                if((c0&0x7f) > (m&0x7f)){
                    r = access_byte(acore, ((c0&0x7f) << 7) + (m&0x7f));
                }else{
                    r = access_byte(acore, ((m&0x7f) << 7) + (c0&0x7f));
                    r ^= 0x80;
                }
            }else{
                // m - c0
                if((m&0x7f) > (c0&0x7f)){
                    r = access_byte(acore, ((m&0x7f) << 7) + (c0&0x7f));
                }else{
                    r = access_byte(acore, ((c0&0x7f) << 7) + (m&0x7f));
                    r ^= 0x80;
                }
            }
        } else {
            if((m&0x7f) <= (c0&0x7f)){
                r = access_byte(acore, ((m&0x7f) << 7) + (c0&0x7f));
            }else{
                r = access_byte(acore, ((c0&0x7f) << 7) + (m&0x7f));
            }
            r |= m & 0x80;
        }
        res |= r << (i * 8);
    }
    return res;
}